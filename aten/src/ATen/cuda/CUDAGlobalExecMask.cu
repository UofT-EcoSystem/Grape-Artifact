// <bojian/DynamicCUDAGraph>

#include "CUDAGlobalExecMask.cuh"
#include "CUDAGlobalExecMask.h"

#include <ATen/core/Formatting.h>
#include <c10/cuda/CUDAStream.h>

#include <dmlc/logging.h>
#include <dmlc/parameter.h>

#include <hip/hip_runtime_api.h>

namespace at {
namespace cuda {

CUDAGlobalExecMask gCUDAGraphGlobalExecMask(true);

std::ostream& operator<<(
    std::ostream& out,
    const CUDAGlobalExecMask& exec_mask) {
  out << "ExecMask{mask=" << exec_mask.mask
      << ", outer_scope_mask=" << exec_mask.outer_scope_mask
      << ", input_ind=" << exec_mask.input_ind << "}";
  return out;
}

CUDAGlobalExecMask::CUDAGlobalExecMask(const bool initialize) {
  if (initialize) {
    checkCudaErrors(hipMalloc(&mask, sizeof(bool)));
    checkCudaErrors(hipMalloc(&outer_scope_mask, sizeof(bool)));
    checkCudaErrors(hipMalloc(&input_ind, sizeof(bool)));
    checkCudaErrors(hipMemset(mask, true, sizeof(bool)));
    checkCudaErrors(hipMemset(outer_scope_mask, true, sizeof(bool)));
    checkCudaErrors(hipMemset(input_ind, true, sizeof(bool)));
  }
}

void EnterGlobalExecMask(
    CUDAGlobalExecMask& outer_scope_exec_mask,
    at::Tensor curr_scope_mask,
    at::Tensor input_ind) {
  // make a copy of the current execution mask
  outer_scope_exec_mask.copy(gCUDAGraphGlobalExecMask);
  gCUDAGraphGlobalExecMask.outer_scope_mask = gCUDAGraphGlobalExecMask.mask;
  gCUDAGraphGlobalExecMask.mask =
      static_cast<bool*>(curr_scope_mask.data_ptr());
  gCUDAGraphGlobalExecMask.input_ind = static_cast<bool*>(input_ind.data_ptr());
}

void ExitGlobalExecMask(CUDAGlobalExecMask& outer_scope_exec_mask) {
  gCUDAGraphGlobalExecMask.copy(outer_scope_exec_mask);
}

} // namespace cuda
} // namespace at
